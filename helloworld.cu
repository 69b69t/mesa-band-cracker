
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

__device__ void setSeed(uint64_t *seed, const uint64_t value)
{
    *seed = (value ^ 0x5deece66d) & ((1ULL << 48) - 1);
}

__device__ int next(uint64_t *seed, const int bits)
{
    *seed = (*seed * 0x5deece66d + 0xb) & ((1ULL << 48) - 1);
    return (int) ((int64_t)*seed >> (48 - bits));
}


//make a lot of hardcoded nextints so ofast can do magic with optimizing out the modulo
__device__ int nextInt(uint64_t *seed, const int n)
{
    int bits, val;
    const int m = n - 1;

    if ((m & n) == 0) {
        uint64_t x = n * (uint64_t)next(seed, 31);
        return (int) ((int64_t) x >> 31);
    }

    do {
        bits = next(seed, 31);
        val = bits % n;
    }
    while (bits - val + m < 0);
    return val;
}

//NEXTINT 3
__device__ int nextInt3(uint64_t *seed)
{
    int bits, val;
    do {
        bits = next(seed, 31);
        val = bits % 3;
    }
    while (bits - val + 2 < 0);
    return val;
}

//NEXTINT 5
__device__ int nextInt5(uint64_t *seed)
{
    int bits, val;
    do {
        bits = next(seed, 31);
        val = bits % 5;
    }
    while (bits - val + 4 < 0);
    return val;
}

//NEXTINT 4
__device__ int nextInt4(uint64_t *seed)
{
    uint64_t x = 4 * (uint64_t)next(seed, 31);
    return (int) ((int64_t) x >> 31);
}

//NEXTINT 4
__device__ int nextInt16(uint64_t *seed)
{
    uint64_t x = 16 * (uint64_t)next(seed, 31);
    return (int) ((int64_t) x >> 31);
}

//NEXTINT 64
__device__ int nextInt64(uint64_t *seed)
{
    uint64_t x = 64 * (uint64_t)next(seed, 31);
    return (int) ((int64_t) x >> 31);
}

__device__ void simplexNoiseStep(uint64_t *seed)
{
    next(seed, 1);
    next(seed, 1);
    next(seed, 1);
    next(seed, 1);
    next(seed, 1);
    next(seed, 1);
    for(int n = 0; n < 256; ++n)
    {
        nextInt(seed, 256 - n);
    }
}

__device__ void printArr(const int arr[64])
{
    for(int i = 0; i < 64; i++)
    {
        printf("%d", arr[i]);
    }
}

__device__ int isViable(const int clayBands[64], const int checkNum)
{
    const int checkAgainst[64] =
    {
        0, 0, 1, 0, 0, 0, 0, 1,
        0, 0, 2, 5, 6, 0, 1, 2,
        0, 0, 1, 0, 0, 0, 1, 0,
        0, 0, 0, 1, 0, 1, 6, -1,
        -1, -1, -1, -1, -1, -1, -1, -1,
        -1, -1, -1, -1, -1, -1, -1, -1,
        -1, -1, -1, -1, -1, -1, -1, -1,
        -1, -1, -1, -1, -1, -1, -1, -1
    };

    for(int i = 0; i < 31; i++)
    {
        if(checkAgainst[i] == checkNum)
        {
            if(checkAgainst[i] != clayBands[i]) return 0;
        }
    }
    return 1;
}

__device__ int nextBoolean(uint64_t *seed)
{
    return next(seed, 1) != 0;
}

__device__ void addOrange(int clayBands[64], uint64_t *seed)
{
    for (int i = 0; i < 64; i++) {
        if ((i += nextInt5(seed) + 1) >= 64) continue;
        clayBands[i] = 1;
    }
}

__device__ void addBrown(int clayBands[64], uint64_t *seed)
{
    int rand1 = nextInt4(seed) + 2;
    int rand2;
    int rand3;
    for (int i = 0; i < rand1; i++) {
        rand2 = nextInt3(seed) + 2;
        rand3 = nextInt64(seed);
        for (int j = 0; rand3 + j < 64 && j < rand2; j++) {
            clayBands[rand3 + j] = 3;
        }
    }
}

__device__ void addColor(int clayBands[64], uint64_t *seed, const int color)
{
    int rand1 = nextInt4(seed) + 2;
    int rand2;
    int rand3;
    for (int i = 0; i < rand1; i++) {
        rand2 = nextInt3(seed) + 1;
        rand3 = nextInt64(seed);
        for (int j = 0; rand3 + j < 64 && j < rand2; j++) {
            clayBands[rand3 + j] = color;
        }
    }
}

__device__ void addWhite(int clayBands[64], uint64_t *seed)
{
    int rand1 = nextInt3(seed) + 3;
    int rand2 = 0;
    for (int i = 0; i < rand1; i++) {
        rand2 += nextInt16(seed) + 4;
        if(rand2 < 64)
        { 
            clayBands[rand2] = 6;
            
            if ((rand2 > 1) && nextBoolean(seed))
            {
                clayBands[rand2 - 1] = 5;
            }
            if ((rand2 < 63) && nextBoolean(seed))
            {
                clayBands[rand2 + 1] = 5;
            }
        }
    }
}

//modulos (nextInt)
// 3 4 5 64

__device__ uint64_t findActualSeed(uint64_t seed)
{
    for(int n = 0; n < 262; n++)
    {
        seed = ((seed - 0xb) * 0xdfe05bcb1365) & ((1ULL << 48) - 1);
    }

    seed = (seed ^ 0x5deece66d);
    return seed;
}

__device__ void generateBands(const uint64_t l)
{
    uint64_t seed;
    seed = l;
    int clayBands[64] = {0};

    //setSeed(&seed, l);

    //simplexNoiseStep(&seed);

    //check correct terracotta per layer
    addOrange(clayBands, &seed);

    //can i remove this?
    //if(isViable(clayBands, 0) == 0) return;
    
    if(!isViable(clayBands, 1)) return;

    addColor(clayBands, &seed, 2);
    //if(!isViable(clayBands, 0)) return;
    //if(!isViable(clayBands, 1)) return;
    if(!isViable(clayBands, 2)) return;

    addBrown(clayBands, &seed);
    //if(!isViable(clayBands, 0)) return;
    //if(!isViable(clayBands, 1)) return;
    //if(!isViable(clayBands, 2)) return;
    if(!isViable(clayBands, 3)) return;

    addColor(clayBands, &seed, 4);
    //if(!isViable(clayBands, 0)) return;
    //if(!isViable(clayBands, 1)) return;
    //if(!isViable(clayBands, 2)) return;
    //if(!isViable(clayBands, 3)) return;
    if(!isViable(clayBands, 4)) return;

    addWhite(clayBands, &seed);
    if(!isViable(clayBands, 0)) return;
    if(!isViable(clayBands, 1)) return;
    if(!isViable(clayBands, 2)) return;
    if(!isViable(clayBands, 3)) return;
    if(!isViable(clayBands, 4)) return;
    if(!isViable(clayBands, 5)) return;
    if(!isViable(clayBands, 6)) return;

    printf("%lld\n", findActualSeed(l));
}

__global__ void craccky(uint64_t bigOffset)
{
    uint64_t idX = bigOffset * 1048576;
    idX += ((uint64_t)threadIdx.x + (uint64_t)blockIdx.x * 1024);
    generateBands(idX);
}

int main()
{
    hipSetDevice(0);
    for(int i = 0; i < 268435456; i++)
    {
        //does 2^20 seeds, out of 2^48, so only(!) need to run it 2^28 times (268435456)
        //to do 2^32, run 2^12 times (4096)
        craccky<<<1024,1024>>>(i);
        hipStreamSynchronize(0);
        if((0x0000ffff & i) == 0) printf("help he voices: (%.14f%%)\n", 100*(i / 268435456.0));
    }
    return 0;
}